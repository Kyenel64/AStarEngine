#include "hip/hip_runtime.h"
﻿#include "RayTracer.cuh"

#include "hiprand/hiprand_kernel.h"

#include <iostream>
#include <chrono>

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ void write_color(unsigned char* frame, int pixel_index, color pixel_color, int samples_per_pixel);
__device__ color ray_color(const Ray& r, Hittable** world, hiprandState *local_rand_state, Data* data);
__global__ void render(unsigned char* frame, Data* data, Hittable** world, Camera** camera, hiprandState *rand_state);
__global__ void render_init(int max_x, int max_y, hiprandState* rand_state);
__global__ void create_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, Material** d_matList, Data* data);
__global__ void free_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, Material** d_matList, Data* data);

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result)
	{
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

RayTracer::RayTracer(Data* data) : data(data)
{
	int num_pixels = data->image_width * data->image_height;
	frame_size = 3 * num_pixels * sizeof(float);
	blockX = 8;
	blockY = 8;
	dim3 blocks(data->image_width / blockX + 1, data->image_height / blockY + 1);
	dim3 threads(blockX, blockY);

	// ------------------ Allocations -----------------------
	checkCudaErrors(hipMalloc(&d_list, data->objectCount * sizeof(Hittable *)));
	checkCudaErrors(hipMalloc(&d_world, sizeof(Hittable *)));
	checkCudaErrors(hipMalloc(&d_data, sizeof(Data)));
	checkCudaErrors(hipMemcpy(d_data, data, sizeof(Data), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc(&d_camera, sizeof(Camera)));
	checkCudaErrors(hipMalloc(&d_matList, data->materialCount * sizeof(Material *)));

	checkCudaErrors(hipMallocManaged(&frame, frame_size));
	checkCudaErrors(hipMallocManaged(&d_rand_state, num_pixels * sizeof(hiprandState)));

	// ------------------- Kernel calls ---------------------
	create_world CUDA_KERNEL(1, 1)(d_list, d_world, d_camera, d_matList, d_data);
	checkCudaErrors(hipDeviceSynchronize());
	render_init CUDA_KERNEL(blocks, threads)(data->image_width, data->image_height, d_rand_state);
	checkCudaErrors(hipDeviceSynchronize());

}

RayTracer::~RayTracer()
{
	free_world CUDA_KERNEL(1, 1)(d_list, d_world, d_camera, d_matList, d_data);
	checkCudaErrors(hipFree(frame));
	checkCudaErrors(hipFree(d_list));
	checkCudaErrors(hipFree(d_world));
	checkCudaErrors(hipFree(d_data));
	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_matList));
	
}

// Returns final rendered frame
unsigned char* RayTracer::getFrame() const
{
	return frame;
}

Data* RayTracer::getData() const
{
	return data;
}

// fills frame[] with render. Acts like main()
bool RayTracer::GenerateFrame()
{
	auto start = std::chrono::high_resolution_clock::now();


	dim3 blocks(data->image_width / blockX + 1, data->image_height / blockY + 1);
	dim3 threads(blockX, blockY);

	render CUDA_KERNEL(blocks, threads)(frame, d_data, d_world, d_camera, d_rand_state);
	checkCudaErrors(hipDeviceSynchronize());


	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
	std::cerr << "\rFinished in: " << duration.count() / 1000.0 << "ms" << std::flush;

	return true;
}

// Write color to array
__device__ void write_color(unsigned char* frame, int pixel_index, color pixel_color, int samples_per_pixel)
{
	float r = pixel_color.r();
	float g = pixel_color.g();
	float b = pixel_color.b();

	// Divide color by number of samples. Gamma correct.
	float scale = 1.0 / samples_per_pixel;
	r = sqrtf(scale * r);
	g = sqrtf(scale * g);
	b = sqrtf(scale * b);

	frame[pixel_index + 0] = int(256 * clamp(r, 0.0, 0.999));
	frame[pixel_index + 1] = int(256 * clamp(g, 0.0, 0.999));
	frame[pixel_index + 2] = int(256 * clamp(b, 0.0, 0.999));
}

// Return color of pixel
__device__ color ray_color(const Ray& r, Hittable **world, hiprandState *local_rand_state, Data* data)
{
	Ray cur_ray = r;
	vec3 cur_attenuation = vec3(1, 1, 1);
	for (int i = 0; i < data->max_depth; i++)
	{
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec))
		{
			Ray scattered;
			vec3 atteuation;
			if (rec.mat_ptr->scatter(cur_ray, rec, atteuation, scattered, local_rand_state))
			{
				cur_attenuation *= atteuation;
				cur_ray = scattered;
			}
			else
			{
				return vec3(0, 0, 0);
			}
		}
		else
		{
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f * (unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void render(unsigned char* frame, Data* data, Hittable **world, Camera **camera, hiprandState *rand_state) {
	// Initializations
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= data->image_width) || (j >= data->image_height))
		return;
	int pixel_index = j * data->image_width * 3 + i * 3;
	int rand_index = j * data->image_width + i;
	hiprandState local_rand_state = rand_state[rand_index];

	color pixel_color;
	for (int s = 0; s < data->samples_per_pixel; s++)
	{
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(data->image_width - 1);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(data->image_height - 1);
		Ray r = (*camera)->get_ray(u, v, &local_rand_state);
		pixel_color += ray_color(r, world, &local_rand_state, data);
	}
	write_color(frame, pixel_index, pixel_color, data->samples_per_pixel);
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state)
{
	// x index and y index
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y))
		return;
	int pixel_index = j * max_x + i;

	// Retrieve a random value for each thread
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

// Allocate world
__global__ void create_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, Material** d_matList, Data* data)
{
	// Allocate new objects and world
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		for (int i = 0; i < data->materialCount; i++)
		{
			if (data->matData[i].matType == 0) // lambert
				d_matList[i] = new Lambertian(data->matData[i].Col, data->matData[i].id);
			if (data->matData[i].matType == 1) // metal
				d_matList[i] = new Lambertian(data->matData[i].Col, data->matData[i].id);
			if (data->matData[i].matType == 2) // dielectric
				d_matList[i] = new Lambertian(data->matData[i].Col, data->matData[i].id);
		}

		for (int i = 0; i < data->objectCount; i++)
		{
			d_list[i] = new Sphere(data->objData[i].Pos, data->objData[i].radius, data->objData[i].id, data->objData[i].matID);
			// Set mat_ptr for sphere to its assigned material.
			for (int j = 0; j < data->materialCount; j++)
			{
				if (d_matList[j]->getID() == d_list[i]->getMatID(data->objData[i].id))
					d_list[i]->mat_ptr = d_matList[j];
			}
		}
		*d_world = new Hittable_list(d_list, data->objectCount);
		*d_camera = new Camera(data);
	}
}

// Deallocate world
__global__ void free_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, Material** d_matList, Data* data)
{
	for (int i = 0; i < data->objectCount; i++)
	{
		delete d_list[i];
	}

	for (int i = 0; i < data->materialCount; i++)
	{
		delete d_matList[i];
	}
	delete* d_world;
	delete* d_camera;
}

__global__ void testKernel(Hittable **world)
{
	(*world)->setPosition(vec3(1, 0, -2));
}

void RayTracer::test()
{
	testKernel CUDA_KERNEL(1, 1)(d_world);
	checkCudaErrors(hipDeviceSynchronize());
}

__global__ void saveKernel(Hittable** world, Material** matList, Data* data)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		for (int i = 0; i < data->objectCount; i++)
		{
			data->objData[i].id = (*world)->getID(i);
			data->objData[i].Pos = (*world)->getPosition(i);
			data->objData[i].radius = (*world)->getRadius(i);
			data->objData[i].matID = (*world)->getMatID(i);
		};

		for (int i = 0; i < data->materialCount; i++)
		{
			data->matData[i].id = matList[i]->getID();
			data->matData[i].Col = matList[i]->getCol();

			if (matList[i]->getType() == lambertian)
				data->matData[i].matType = matList[i]->getType();
			
		}
	}
	
}

void RayTracer::save()
{
	saveKernel CUDA_KERNEL(1, 1)(d_world, d_matList, d_data);
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(data, d_data, sizeof(Data), hipMemcpyDeviceToHost));
}

__global__ void addObjectKernel(Hittable** d_list, Hittable** d_world, Material** d_matList, Data* data, vec3 Pos, float radius)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		
	}
}

void RayTracer::addObject(vec3 Pos, float radius)
{
	
}