#include "hip/hip_runtime.h"
﻿#include "RayTracer.cuh"
#include "Classes/vec3.cuh"
#include "Classes/ray.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

RayTracer::RayTracer(int width, int height) : width(width), height(height)
{
	int num_pixels = width * height;
	frame_size = 3 * num_pixels * sizeof(float);
	hipMallocManaged(&frame, frame_size);
	blockX = 4;
	blockY = 4;

}

RayTracer::~RayTracer()
{
	hipFree(frame);
}

// Returns frame to render to texture
unsigned char* RayTracer::getFrame() const
{
	return frame;
}

__device__ void write_color(unsigned char* frame, int pixel_index, color pixel_color)
{
	frame[pixel_index + 0] = int(255.99 * (pixel_color.r()));
	frame[pixel_index + 1] = int(255.99 * (pixel_color.g()));
	frame[pixel_index + 2] = int(255.99 * (pixel_color.b()));
}

__device__ color ray_color(const Ray& r)
{
	vec3 unit_direction = unit_vector(r.direction());
	float t = 0.5 * (unit_direction.y() + 1.0);
	return (1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0);
}

__global__ void render(unsigned char* frame, int max_x, int max_y) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x * 3 + i * 3;

	color pixel_color(float(i) / max_x, float(j) / max_y, 0.2);
	write_color(frame, pixel_index, pixel_color);
}

// fills frame[] with render. Acts like main()
bool RayTracer::GenerateFrame(double time)
{
	dim3 blocks(width / blockX + 1, height / blockY + 1);
	dim3 threads(blockX, blockY);
	render CUDA_KERNEL(blocks, threads)(frame, width, height);
	hipDeviceSynchronize();


	return true;
}

