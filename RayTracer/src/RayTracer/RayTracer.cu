#include "hip/hip_runtime.h"
﻿#include "RayTracer.cuh"

#include "hiprand/hiprand_kernel.h"

#include <iostream>
#include <chrono>

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

__device__ void write_color(unsigned char* frame, int pixel_index, color pixel_color, int samples_per_pixel);
__device__ color ray_color(const Ray& r, Hittable** world, hiprandState *local_rand_state, Data* data);
__global__ void render(unsigned char* frame, Data* data, Hittable** world, Camera** camera, hiprandState *rand_state);
__global__ void create_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, Material** d_matList, Data* data);
__global__ void free_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, Material** d_matList, Data* data);
__global__ void render_init(int max_x, int max_y, hiprandState* rand_state);

RayTracer::RayTracer(Data* data) : data(data)
{
	int num_pixels = data->image_width * data->image_height;
	frame_size = 3 * num_pixels * sizeof(float);
	blockX = 8;
	blockY = 8;

	// ------------------ Allocations -----------------------
	hipMalloc(&d_list, data->objectCount * sizeof(Hittable *));
	hipMalloc(&d_world, sizeof(Hittable *));
	hipMalloc(&d_data, sizeof(Data));
	hipMemcpy(d_data, data, sizeof(Data), hipMemcpyHostToDevice);
	hipMalloc(&d_camera, sizeof(Camera));
	hipMalloc(&d_matList, data->materialCount * sizeof(Material *));

	hipMallocManaged(&frame, frame_size);
	hipMallocManaged(&d_rand_state, num_pixels * sizeof(hiprandState));
	dim3 blocks(data->image_width / blockX + 1, data->image_height / blockY + 1);
	dim3 threads(blockX, blockY);

	// ------------------- Kernel calls ---------------------
	create_world CUDA_KERNEL(1, 1)(d_list, d_world, d_camera, d_matList, d_data);
	hipDeviceSynchronize();
	render_init CUDA_KERNEL(blocks, threads)(data->image_width, data->image_height, d_rand_state);
	hipDeviceSynchronize();

}

RayTracer::~RayTracer()
{
	free_world CUDA_KERNEL(1, 1)(d_list, d_world, d_camera, d_matList, d_data);
	hipFree(frame);
	hipFree(d_list);
	hipFree(d_world);
	hipFree(d_data);
	hipFree(d_camera);
	hipFree(d_matList);
	
}

// Returns final rendered frame
unsigned char* RayTracer::getFrame() const
{
	return frame;
}

Data* RayTracer::getData() const
{
	return data;
}

// fills frame[] with render. Acts like main()
bool RayTracer::GenerateFrame()
{
	auto start = std::chrono::high_resolution_clock::now();


	dim3 blocks(data->image_width / blockX + 1, data->image_height / blockY + 1);
	dim3 threads(blockX, blockY);

	render CUDA_KERNEL(blocks, threads)(frame, d_data, d_world, d_camera, d_rand_state);
	hipDeviceSynchronize();


	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
	std::cerr << "\rFinished in: " << duration.count() / 1000.0 << "ms" << std::flush;

	return true;
}

// Write color to array
__device__ void write_color(unsigned char* frame, int pixel_index, color pixel_color, int samples_per_pixel)
{
	float r = pixel_color.r();
	float g = pixel_color.g();
	float b = pixel_color.b();

	// Divide color by number of samples. Gamma correct.
	float scale = 1.0 / samples_per_pixel;
	r = sqrtf(scale * r);
	g = sqrtf(scale * g);
	b = sqrtf(scale * b);

	frame[pixel_index + 0] = int(256 * clamp(r, 0.0, 0.999));
	frame[pixel_index + 1] = int(256 * clamp(g, 0.0, 0.999));
	frame[pixel_index + 2] = int(256 * clamp(b, 0.0, 0.999));
}

// Return color of pixel
__device__ color ray_color(const Ray& r, Hittable **world, hiprandState *local_rand_state, Data* data)
{
	Ray cur_ray = r;
	vec3 cur_attenuation = vec3(1, 1, 1);
	for (int i = 0; i < data->max_depth; i++)
	{
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec))
		{
			Ray scattered;
			vec3 atteuation;
			if (rec.mat_ptr->scatter(cur_ray, rec, atteuation, scattered, local_rand_state))
			{
				cur_attenuation *= atteuation;
				cur_ray = scattered;
			}
			else
			{
				return vec3(0, 0, 0);
			}
		}
		else
		{
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f * (unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void render(unsigned char* frame, Data* data, Hittable **world, Camera **camera, hiprandState *rand_state) {
	// Initializations
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= data->image_width) || (j >= data->image_height))
		return;
	int pixel_index = j * data->image_width * 3 + i * 3;
	int rand_index = j * data->image_width + i;
	hiprandState local_rand_state = rand_state[rand_index];

	color pixel_color;
	for (int s = 0; s < data->samples_per_pixel; s++)
	{
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(data->image_width - 1);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(data->image_height - 1);
		Ray r = (*camera)->get_ray(u, v);
		pixel_color += ray_color(r, world, &local_rand_state, data);
	}
	write_color(frame, pixel_index, pixel_color, data->samples_per_pixel);
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state)
{
	// x index and y index
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y))
		return;
	int pixel_index = j * max_x + i;

	// Retrieve a random value for each thread
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

// Allocate world
__global__ void create_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, Material** d_matList, Data* data)
{
	// Allocate new objects and world
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		for (int i = 0; i < data->materialCount; i++)
		{
			if (data->matData[i].matType == 0) // lambert
				d_matList[i] = new Lambertian(data->matData[i].Col, data->matData[i].id);
			if (data->matData[i].matType == 1) // metal
				d_matList[i] = new Lambertian(data->matData[i].Col, data->matData[i].id);
			if (data->matData[i].matType == 2) // dielectric
				d_matList[i] = new Lambertian(data->matData[i].Col, data->matData[i].id);
		}

		for (int i = 0; i < data->objectCount; i++)
		{
			d_list[i] = new Sphere(data->objData[i].Pos, data->objData[i].radius, data->objData[i].id, data->objData[i].matID, d_matList[data->objData[i].matID]);
		}
		*d_world = new Hittable_list(d_list, data->objectCount);
		*d_camera = new Camera();
	}
}

// Deallocate world
__global__ void free_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, Material** d_matList, Data* data)
{
	for (int i = 0; i < data->objectCount; i++)
	{
		delete d_list[i];
	}

	for (int i = 0; i < data->materialCount; i++)
	{
		delete d_matList[i];
	}

	//delete ((Sphere*)d_list[0])->mat_ptr;
	delete* d_world;
	delete* d_camera;
}

__global__ void testKernel(Hittable **world)
{
	(*world)->setPosition(vec3(1, 0, -2));
}

void RayTracer::test()
{
	testKernel CUDA_KERNEL(1, 1)(d_world);
	hipDeviceSynchronize();
}

__global__ void saveKernel(Hittable** world, Material** matList, Data* data)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		for (int i = 0; i < data->objectCount; i++)
		{
			data->objData[i].id = (*world)->getID(i);
			data->objData[i].Pos = (*world)->getPosition(i);
			data->objData[i].radius = (*world)->getRadius(i);
			data->objData[i].matID = (*world)->getMatID(i);
		};

		for (int i = 0; i < data->materialCount; i++)
		{
			if (matList[i]->getType() == lambertian)
			{
				data->matData[i].id = matList[i]->getID();
				data->matData[i].Col = matList[i]->getCol();
				data->matData[i].matType = matList[i]->getType();
			}
			
		}
	}
	
}

void RayTracer::save()
{
	saveKernel CUDA_KERNEL(1, 1)(d_world, d_matList, d_data);
	hipDeviceSynchronize();
	hipMemcpy(data, d_data, sizeof(Data), hipMemcpyDeviceToHost);
}

//__global__ void addObjectKernel(Hittable** d_list, Hittable** d_world, Material** d_matList, Data* data, vec3 Pos, float radius)
//{
//	if (threadIdx.x == 0 && blockIdx.x == 0)
//	{
//		delete* d_world;
//		*d_world = new Hittable_list(d_list, data->objectCount + 1);
//		d_matList[data->materialCount] = new Lambertian(color(1.0, 0.5, 0.5), data->materialCount);
//		d_list[data->objectCount] = new Sphere(Pos, radius, data->objectCount, data->materialCount, d_matList[data->materialCount]);
//		data->objectCount++;
//		data->materialCount++;
//	}
//}
//
//void RayTracer::addObject(int id, vec3 Pos, float radius)
//{
//	hipFree(d_list);
//	hipMalloc(&d_list, data->objectCount+1 * sizeof(Hittable*));
//	hipFree(d_matList);
//	hipMalloc(&d_matList, data->materialCount + 1 * sizeof(Material*));
//	addObjectKernel CUDA_KERNEL(1, 1)(d_list, d_world, d_matList, d_data, Pos, radius);
//	hipDeviceSynchronize();
//	hipMemcpy(data, d_data, sizeof(Data), hipMemcpyDeviceToHost);
//}